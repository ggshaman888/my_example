#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "particle.h"
#include <stdlib.h>
#include <stdio.h>
# include <iostream>
# include <vector>
# include <fstream>
# include <numeric>
# include <cmath>
# include <chrono>
# include "sparsematrix.h"

const std::size_t Max_Iter = 50;

void print(const std::vector<double> &X);
std::vector<double> solve_SI(
    const SparseMatrix &A,
    double Epsilon);

std::vector<double> multiple(
    SparseMatrix &matrix,
    const std::vector<double> &vec);
double norm(const std::vector<double> &vec);
std::vector<double> plus(
    const std::vector<double> &first,
    const std::vector<double> &second);
std::vector<double> multiple(
    const std::vector<double> &vec,
    const double scalar);
void TransformMatrix(SparseMatrix &matrix);

//__global__ void advanceParticles(float dt, particle * pArray, int nParticles)
//{
//    int idx = threadIdx.x + blockIdx.x*blockDim.x; // number thread
//	if(idx < nParticles)
//	{
//		pArray[idx].advance(dt);
//	}
//}
__global__ void advanceParticles(particle * pArray, int nParticles, int nrows)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x; // number thread
    if(idx < nParticles)
    {

      //  pArray[idx].advance(dt);
        for(int j = 0; j < nrows; j++)
        {
            double x = pArray->matrixS.get(idx,j);
            double y = pArray->vecS[j];
            double r = pArray->Res[idx];
            double z = x*y;
            r = r+z;
            pArray->Res[idx] =r;
        }
    }
}
std::vector<double> multiple(
        SparseMatrix &matrix,
        const std::vector<double> &vec)
{
    std::vector<double> Res(0);
    hipError_t error;
  //  int n = 1000000;
    int n = vec.size();
//    if(argc > 1)	{ n = atoi(argv[1]);}     // Number of particles
//    if(argc > 2)	{	srand(atoi(argv[2])); } // Random seed

    error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("0 %s\n",hipGetErrorString(error));
    exit(1);
    }

    particle * pArray = new particle[0];
    pArray->multiple(matrix, vec);
    particle * devPArray = NULL;
    hipMalloc(&devPArray, sizeof(pArray));
    hipDeviceSynchronize(); error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("1 %s\n",hipGetErrorString(error));
    exit(1);
    }

    hipMemcpy(devPArray, pArray, sizeof(pArray), hipMemcpyHostToDevice);
    hipDeviceSynchronize(); error = hipGetLastError();
    if (error != hipSuccess)
    {
    printf("2 %s\n",hipGetErrorString(error));
    exit(1);
    }
        int bufN = pArray->vecS.size();
        int buf_nrows = pArray->matrixS.m_nrows;
    //for(int i=0; i<100; i++)
    //{
//        float dt = (float)rand()/(float) RAND_MAX; // Random distance each step
        advanceParticles<<< 1 +  n/256, 256>>>(devPArray, bufN, buf_nrows);
        error = hipGetLastError();
        if (error != hipSuccess)
        {
        printf("3 %s\n",hipGetErrorString(error));
        exit(1);
        }

        hipDeviceSynchronize();
    //}
    hipMemcpy(pArray, devPArray, sizeof(pArray), hipMemcpyDeviceToHost);

//    v3 totalDistance(0,0,0);
//    v3 temp;
//    for(int i=0; i<n; i++)
//    {
//        temp = pArray[i].getTotalDistance();
//        totalDistance.x += temp.x;
//        totalDistance.y += temp.y;
//        totalDistance.z += temp.z;
//    }
//    float avgX = totalDistance.x /(float)n;
//    float avgY = totalDistance.y /(float)n;
//    float avgZ = totalDistance.z /(float)n;
//    float avgNorm = sqrt(avgX*avgX + avgY*avgY + avgZ*avgZ);
//    printf(	"Moved %d particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n",
//                    n, avgX, avgY, avgZ, avgNorm);


    return pArray->Res;
}
int main(int argc, char ** argv)
{
    auto matrix = SparseMatrix("a01.txt","b01.txt");
    auto start = std::chrono::steady_clock::now();
    auto X = solve_SI(matrix,1.0e-020);





    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    std::cout <<"Task takes "<< std::chrono::duration_cast<std::chrono::seconds>(diff).count() << " seconds\n";
    std::ofstream out("out_SI.txt");
    for(const auto &el : X)
        out<<el<<'\n';
    return 0;
}

//вычисление нормы
double norm(const std::vector<double> &vec)
{
    return sqrt(std::accumulate(vec.begin(),vec.end(),0.0,[](double x,double y)
    {
        return x+y*y;
    }));
}
//сложение векторов
std::vector<double> plus(
    const std::vector<double> &first,
    const std::vector<double> &second)
{
    std::vector<double> r(first.size(),0);
    std::transform(first.begin(),first.end(),second.begin(),r.begin(),[](const double &x,const double &y)
    {
        return x+y;
    });
    return r;
}
//умножение вектора на скаляр (поэлементное)
std::vector<double> multiple(
    const std::vector<double> &vec,
    const double scalar)
{
    std::vector<double> R(vec.size());
    std::transform(vec.begin(),vec.end(),R.begin(),[&scalar](const double &x)
    {
        return x*scalar;
    });
    return R;
}
//приведение матрицы к необходимому виду
void TransformMatrix(SparseMatrix &matrix)
{
    auto n = matrix.m_nrows;
    for(std::size_t i=0; i<n; i++)
    {
        auto el = matrix.get(i,i);
        if(el>0)
        {
            matrix.set(1-el,i,i);
            for(size_t j=0; j<n; j++)
                if(i!=j)
                    matrix.set(-matrix.get(i,j),i,j);
        }
        else
        {
            matrix.set(1+el,i,i);
            matrix.set(-matrix.get(i,n),i,n);
        }
    }
}
//реализация метода
std::vector<double> solve_SI(
    const SparseMatrix &A,
    double Epsilon)
{
    std::size_t iter=1;
    auto Matrix = A;
    auto n = Matrix.m_nrows;

    if(n == 0)
        return std::vector<double>{};

    auto B = Matrix.get_column(n);

    TransformMatrix(Matrix);
    std::cout << "start" << std::endl;
    auto X = std::vector<double>(n,0);
    while(iter<Max_Iter)
    {
        //делаем шаг
        auto Xnew = plus(multiple(Matrix,X),B);
        //расчитываем норму приращения(разница между 2-мя приближениями)
        double n2 = norm(plus(Xnew,multiple(X,-1)));

        if(n2<Epsilon)
        {
            std::cout<<"Количество итераций:"<<iter<<'\n';
            return X;
        }
        X = Xnew;
        ++iter;
    }
    std::cout<<"Метод расходится\n";
    return X;
}

void print(const std::vector<double> &X)
{
    for(const auto &el : X)
        std::cout<<el<<' ';
    std::cout<<'\n';
}
